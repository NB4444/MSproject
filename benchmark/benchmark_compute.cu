#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void kernel(float* A, long const num_runs) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    float a = 0;
    for(long i =0; i < num_runs; i++) a += 1.234;
    A[index] = a;
}

int main(int argc, char **argv) {
    int num_sms = atoi(argv[1]);
    int num_threads = atoi(argv[2]);
    long const num_runs = atol(argv[3]);
    int total = num_sms * num_threads;

    float* A = NULL;
    hipMalloc((void **) &A, total * sizeof(float));

    kernel<<<num_sms, num_threads>>>(A, num_runs);

    hipFree(A);
}