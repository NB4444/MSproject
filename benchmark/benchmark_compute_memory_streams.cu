#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void kernel(float* A, float* B, long const num_runs) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    for(long i =0; i < num_runs; i++) A[index] += B[index];
}
int main(int argc, char **argv) {
    int num_sms = atoi(argv[1]);
    int num_threads = atoi(argv[2]);
    long const num_runs = atol(argv[3]);
    int num_streams = atoi(argv[4]);
    int total = num_sms * num_threads;

    hipStream_t streams[num_streams];
    for (int i = 0; i < num_streams; i++) hipStreamCreate(&(streams[i]));

    float* A = NULL;
    hipMalloc((void **) &A, total * sizeof(float) * num_streams);

    float* B = NULL;
    hipMalloc((void **) &B, total * sizeof(float) * num_streams);

    for (int i = 0; i < num_streams; i++) kernel<<<num_sms, num_threads, 0, streams[i]>>>(&(A[total*i]), &(B[total*i]), num_runs);

    hipFree(A);
}