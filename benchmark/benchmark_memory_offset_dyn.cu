#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        printf("cuda error \n");
        exit(1);
    }
}

__global__ void kernel(float* A, float* B, long const num_runs, int s, int total) {
    int index = (blockIdx.x * blockDim.x + threadIdx.x) * s;
    int sq = s * s;
    long size = (num_runs*sq)+index;
    int array_size = total * sq;
    for(long i = index; i < size; i+=sq) {
        int new_index = i % array_size;
        A[new_index] = B[new_index];
    }
}

int main(int argc, char **argv) {
    int num_sms = atoi(argv[1]);
    int num_threads = atoi(argv[2]);
    long const num_runs = atol(argv[3]);
    int s = atoi(argv[4]);
    int total = num_sms * num_threads;


    float* A = NULL;
    checkCudaCall(hipMalloc((void **) &A, total * s * s *  sizeof(float)));

    float* B = NULL;
    checkCudaCall(hipMalloc((void **) &B, total * s * s * sizeof(float)));

    kernel<<<num_sms, num_threads>>>(A, B, num_runs, s, total);

    hipFree(A);
}