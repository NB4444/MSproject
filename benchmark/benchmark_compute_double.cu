#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void kernel(double* A, int total, long const num_runs) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    double a = 0;
    for(long i =0; i < num_runs; i++) a += 1.234;
    A[index] = a;
}

int main(int argc, char **argv) {
    int num_sms = atoi(argv[1]);
    int num_threads = atoi(argv[2]);
    long const num_runs = atol(argv[3]);
    int total = num_sms * num_threads;

    double* A = NULL;
    hipMalloc((void **) &A, total * sizeof(float));

    kernel<<<num_sms, num_threads>>>(A, total, num_runs);

    hipFree(A);
}